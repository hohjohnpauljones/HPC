
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#include <cstdio>
#include <cstdlib>

typedef unsigned char uint8_t;

int main (int argc, char *argv[]) {

    if (argc != 3) // Change me per specs
        return 1;

    int height, width;
    char magic_number[4], input[10];
    int gray_scale;

    //Reads from argv[1] the input pgm file
    FILE *fp = fopen(argv[1],"r");
    fgets(magic_number, 4, fp);
    magic_number[2] = '\0';
    fgets(input, 10, fp);
    height = atoi(input);
    fgets(input, 10, fp);
    width = atoi(input);
    fgets(input, 10, fp);
    gray_scale = atoi(input);

    std::vector<uint8_t> mat(height * width);
    //Populates the arrays
    for (int i= 0; i < height * width; i++)
        mat[i] = fgetc(fp);

    fclose(fp);

    std::vector<uint8_t> median(height * width);
    uint8_t *d_input, *d_output;
    hipMalloc((void **) &d_input, height * width * sizeof(uint8_t));
    hipMalloc((void **) &d_output, height * width * sizeof(uint8_t));
    hipMemcpy(d_input, &mat[0], height * width * sizeof(uint8_t), hipMemcpyHostToDevice);

    // TODO - Fill median.
	
	for (int i = 0; i < height * width; i++)
		median.push_back(mat[i]);
		//median[i] = mat[i];
	
    hipMemcpy(&median[0], d_output, height * width * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);

    //Writes the new pgm picture
    fp = fopen(argv[2], "w");
    fprintf(fp, "%s\n%d\n%d\n%d\n", magic_number, height, width, gray_scale);
    for (int i=0;i<median.size();i++)
        fputc(median[i], fp);
    fclose(fp);

    return 0;
}
