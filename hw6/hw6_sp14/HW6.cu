
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#include <cstdio>
#include <cstdlib>

typedef unsigned char uint8_t;

__global__ void kernel( uint8_t *d_input, uint8_t *d_output) {
	// map from threadIdx/BlockIdx to pixel position
	int x = blockIdx.x;
	int y = blockIdx.y;
	int dim = 3;
	//int mid = dim / 2 + 1;
	//int offset = x + y * gridDim.x;
	//int offset2 = offset;
	//offset2 = x + (gridDim.y - y);
	//offset2 = y + x * gridDim.y;
	//offset2 = y + (gridDim.x * (gridDim.y - x - 1));
	
	//d_output[offset] = d_input[offset2];

	const int yOffset = y * gridDim.x;
	const int yPrev = yOffset - gridDim.x;
	const int yNext = yOffset + gridDim.x;
	
	float neighborhood[9];
	
	
	if (y > 0 && y < (gridDim.y - 1) && x > 0 && x < (gridDim.x - 1))
	{

        	neighborhood[0] = d_input[yPrev + x - 1];
        	neighborhood[1] = d_input[yPrev + x];
        	neighborhood[2] = d_input[yPrev + x + 1];
        	neighborhood[3] = d_input[yOffset + x - 1];

        	neighborhood[4] = d_input[yOffset + x];

        	neighborhood[5] = d_input[yOffset + x + 1];
        	neighborhood[6] = d_input[yNext + x - 1];
        	neighborhood[7] = d_input[yNext + x];
        	neighborhood[8] = d_input[yNext + x + 1];
	}
	else
	{
		neighborhood[0] = 0;
                neighborhood[1] = 0;
                neighborhood[2] = 0;
                neighborhood[3] = 0;

                neighborhood[4] = d_input[yOffset + x];

                neighborhood[5] = 255;
                neighborhood[6] = 255;
                neighborhood[7] = 255;
                neighborhood[8] = 255;
	}

	//sort neighborhood
	
	for (int i = 0; i < 8; i++)
	{
		for (int j = i; j < 8; j++)
		{
			if (neighborhood[i] > neighborhood[i + 1])
			{
				int temp = neighborhood[i];
				neighborhood[i] = neighborhood[i+1];
				neighborhood[i+1] = temp;
			}
		}
	}
	
	// assign pixel to median

	d_output[yOffset + x] = neighborhood[5];

}

int main (int argc, char *argv[]) {

    if (argc != 3) // Change me per specs
        return 1;

    int height, width;
    char magic_number[4], input[10];
    int gray_scale;

    //Reads from argv[1] the input pgm file
    FILE *fp = fopen(argv[1],"r");
    fgets(magic_number, 4, fp);
    magic_number[2] = '\0';
	//read up to 10 characters or new line
    fgets(input, 10, fp);
    height = atoi(input);
    fgets(input, 10, fp);
    width = atoi(input);
    fgets(input, 10, fp);
    gray_scale = atoi(input);

    std::vector<uint8_t> mat(height * width);
    //Populates the arrays grabing each pixel from the image and storing it into the vector.
    for (int i= 0; i < height * width; i++)
        mat[i] = fgetc(fp);

    fclose(fp);

    std::vector<uint8_t> median(height * width);
    uint8_t *d_input, *d_output;
    hipMalloc((void **) &d_input, height * width * sizeof(uint8_t));
    hipMalloc((void **) &d_output, height * width * sizeof(uint8_t));
	//copy the image that we read, into d_input and send it over to the GPU's memory
    hipMemcpy(d_input, &mat[0], height * width * sizeof(uint8_t), hipMemcpyHostToDevice);

    // TODO - Fill median.
	/*
	for (int i = 0; i < height * width; i++)
		d_output[i] = d_input[i];
		//median.push_back(d_input[i]);
		//median[i] = mat[i];
	*/
	dim3 grid(height, width);

	kernel<<<grid,1>>>(d_input, d_output);
    hipMemcpy(&median[0], d_output, height * width * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);

    //Writes the new pgm picture
    fp = fopen(argv[2], "w");
    fprintf(fp, "%s\n%d\n%d\n%d\n", magic_number, height, width, gray_scale);
    for (int i=0;i<median.size();i++)
        fputc(median[i], fp);
    fclose(fp);

    return 0;
}

