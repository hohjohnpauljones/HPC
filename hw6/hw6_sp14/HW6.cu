
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#include <cstdio>
#include <cstdlib>

typedef unsigned char uint8_t;

/* This function swaps two numbers
   Arguments :
			 a, b - the numbers to be swapped
   */
__device__ void swap(uint8_t &a, uint8_t &b)
{
	int temp;
	temp = a;
	a = b;
	b = temp;
}

/* This function splits the array around the pivot
   Arguments :
			 array - the array to be split
			 pivot - pivot element whose position will be returned
			 startIndex - index of the first element of the section
			 endIndex - index of the last element of the section
   Returns :
		   the position of the pivot
   */
__device__ int SplitArray(uint8_t* array, int pivot, int startIndex, int endIndex)
{
	int leftBoundary = startIndex;
	int rightBoundary = endIndex;
	
	while(leftBoundary < rightBoundary)			   //shuttle pivot until the boundaries meet
	{
		 while( pivot < array[rightBoundary]		  //keep moving until a lesser element is found
				&& rightBoundary > leftBoundary)	  //or until the leftBoundary is reached
		 {
			  rightBoundary--;						//move left
		 }
		 swap(array[leftBoundary], array[rightBoundary]);
		 
		 while( pivot >= array[leftBoundary]		  //keep moving until a greater or equal element is found
				&& leftBoundary < rightBoundary)	  //or until the rightBoundary is reached
		 {
			  leftBoundary++;						 //move right
		 }
		 swap(array[rightBoundary], array[leftBoundary]);
	}
	return leftBoundary;
}

/* This function does the quicksort
   Arguments :
			 array - the array to be sorted
			 startIndex - index of the first element of the section
			 endIndex - index of the last element of the section
   */
__device__ void QuickSort(uint8_t* array, int startIndex, int endIndex)
{
	int pivot = array[startIndex];	//pivot element is the leftmost element
	int splitPoint;
	
	if(endIndex > startIndex)
	{
		splitPoint = SplitArray(array, pivot, startIndex, endIndex);
		array[splitPoint] = pivot;
		QuickSort(array, startIndex, splitPoint-1);   //Quick sort first half
		QuickSort(array, splitPoint+1, endIndex);	 //Quick sort second half
	}
}

__global__ void medianFilter3( uint8_t *d_input, uint8_t *d_output) {
        // map from threadIdx/BlockIdx to pixel position^M
        int x = blockIdx.x;
        int y = blockIdx.y;
        int dim = 3;

	const int yOffset = y * gridDim.x;
	const int yPrev = yOffset - gridDim.x;
	const int yNext = yOffset + gridDim.x;
	
	uint8_t neighborhood[9];
	
	
	if (y > 0 && y < (gridDim.y - 1) && x > 0 && x < (gridDim.x - 1))
	{

        	neighborhood[0] = d_input[yPrev + x - 1];
        	neighborhood[1] = d_input[yPrev + x];
        	neighborhood[2] = d_input[yPrev + x + 1];
        	
        	neighborhood[3] = d_input[yOffset + x - 1];

        	neighborhood[4] = d_input[yOffset + x];

        	neighborhood[5] = d_input[yOffset + x + 1];
        	
        	neighborhood[6] = d_input[yNext + x - 1];
        	neighborhood[7] = d_input[yNext + x];
        	neighborhood[8] = d_input[yNext + x + 1];
	}
	else
	{
		neighborhood[0] = 0;
		neighborhood[1] = 0;
		neighborhood[2] = 0;
		neighborhood[3] = 0;
		
		neighborhood[4] = d_input[yOffset + x];
		
		neighborhood[5] = 255;
		neighborhood[6] = 255;
		neighborhood[7] = 255;
		neighborhood[8] = 255;
	}

	//sort neighborhood
	QuickSort(neighborhood, 0, 9);
	
	// assign pixel to median

	d_output[yOffset + x] = neighborhood[5];

}

__global__ void medianFilter7( uint8_t *d_input, uint8_t *d_output) {
        // map from threadIdx/BlockIdx to pixel position^M
        int x = blockIdx.x;
        int y = blockIdx.y;
        int dim = 7;
	
	const int yOffset = y * gridDim.x;
	
	const int yOffset1 = yOffset - gridDim.x * 3;
	const int yOffset2 = yOffset - gridDim.x * 2;
	const int yOffset3 = yOffset - gridDim.x * 1;
	const int yOffset5 = yOffset + gridDim.x * 1;
	const int yOffset6 = yOffset + gridDim.x * 2;
	const int yOffset7 = yOffset + gridDim.x * 3;
	
	
	
	int yOffsets[7];
	
	yOffsets[0] = yOffset - gridDim.x * 3;
	yOffsets[1] = yOffset - gridDim.x * 2;
	yOffsets[2] = yOffset - gridDim.x * 1;
	yOffsets[3] = yOffset;
	yOffsets[4] = yOffset + gridDim.x * 1;
	yOffsets[5] = yOffset + gridDim.x * 2;
	yOffsets[6] = yOffset + gridDim.x * 3;
	
	uint8_t neighborhood[7 * 7];
	
	
	if (y > 0 && y < (gridDim.y - 1) && x > 0 && x < (gridDim.x - 1))
	{

        	neighborhood[0] = d_input[yOffset1 + x - 3];
        	neighborhood[1] = d_input[yOffset1 + x - 2];
        	neighborhood[2] = d_input[yOffset1 + x - 1];
        	neighborhood[3] = d_input[yOffset1 + x - 0];
        	neighborhood[4] = d_input[yOffset1 + x + 1];
        	neighborhood[5] = d_input[yOffset1 + x + 2];
        	neighborhood[6] = d_input[yOffset1 + x + 3];
        	
		neighborhood[7] = d_input[yOffset2 + x - 3];
        	neighborhood[8] = d_input[yOffset2 + x - 2];
        	neighborhood[9] = d_input[yOffset2 + x - 1];
        	neighborhood[10] = d_input[yOffset2 + x - 0];
        	neighborhood[11] = d_input[yOffset2 + x + 1];
        	neighborhood[12] = d_input[yOffset2 + x + 2];
        	neighborhood[13] = d_input[yOffset2 + x + 3];
        	
        	neighborhood[14] = d_input[yOffset3 + x - 3];
        	neighborhood[15] = d_input[yOffset3 + x - 2];
        	neighborhood[16] = d_input[yOffset3 + x - 1];
        	neighborhood[17] = d_input[yOffset3 + x - 0];
        	neighborhood[18] = d_input[yOffset3 + x + 1];
        	neighborhood[19] = d_input[yOffset3 + x + 2];
        	neighborhood[20] = d_input[yOffset3 + x + 3];
        	
        	neighborhood[21] = d_input[yOffset + x - 3];
        	neighborhood[22] = d_input[yOffset + x - 2];
        	neighborhood[23] = d_input[yOffset + x - 1];
        	
        	neighborhood[24] = d_input[yOffset + x - 0];
        	
        	neighborhood[25] = d_input[yOffset + x + 1];
        	neighborhood[26] = d_input[yOffset + x + 2];
        	neighborhood[27] = d_input[yOffset + x + 3];
        	
        	neighborhood[28] = d_input[yOffset5 + x - 3];
        	neighborhood[29] = d_input[yOffset5 + x - 2];
        	neighborhood[30] = d_input[yOffset5 + x - 1];
        	neighborhood[31] = d_input[yOffset5 + x - 0];
        	neighborhood[32] = d_input[yOffset5 + x + 1];
        	neighborhood[33] = d_input[yOffset5 + x + 2];
        	neighborhood[34] = d_input[yOffset5 + x + 3];
        	
        	neighborhood[35] = d_input[yOffset6 + x - 3];
        	neighborhood[35] = d_input[yOffset6 + x - 2];
        	neighborhood[37] = d_input[yOffset6 + x - 1];
        	neighborhood[38] = d_input[yOffset6 + x - 0];
        	neighborhood[39] = d_input[yOffset6 + x + 1];
        	neighborhood[40] = d_input[yOffset6 + x + 2];
        	neighborhood[41] = d_input[yOffset6 + x + 3];
        	
        	neighborhood[42] = d_input[yOffset7 + x - 3];
        	neighborhood[43] = d_input[yOffset7 + x - 2];
        	neighborhood[44] = d_input[yOffset7 + x - 1];
        	neighborhood[45] = d_input[yOffset7 + x - 0];
        	neighborhood[46] = d_input[yOffset7 + x + 1];
        	neighborhood[47] = d_input[yOffset7 + x + 2];
        	neighborhood[48] = d_input[yOffset7 + x + 3];
        	
	}
	else
	{
        	neighborhood[0] = 0;
        	neighborhood[1] = 0;
        	neighborhood[2] = 0;
        	neighborhood[3] = 0;
        	neighborhood[4] = 0;
        	neighborhood[5] = 0;
        	neighborhood[6] = 0;
        	
		neighborhood[7] =  0;
        	neighborhood[8] =  0;
        	neighborhood[9] =  0;
        	neighborhood[10] = 0;
        	neighborhood[11] = 0;
        	neighborhood[12] = 0;
        	neighborhood[13] = 0;
        	
        	neighborhood[14] = 0;
        	neighborhood[15] = 0;
        	neighborhood[16] = 0;
        	neighborhood[17] = 0;
        	neighborhood[18] = 0;
        	neighborhood[19] = 0;
        	neighborhood[20] = 0;
        	
        	neighborhood[21] = 0;
        	neighborhood[22] = 0;
        	neighborhood[23] = 0;
        	
        	neighborhood[24] = d_input[yOffset + x - 0];
        	
        	neighborhood[25] = 255;
        	neighborhood[26] = 255;
        	neighborhood[27] = 255;
        	
        	neighborhood[28] = 255;
        	neighborhood[29] = 255;
        	neighborhood[30] = 255;
        	neighborhood[31] = 255;
        	neighborhood[32] = 255;
        	neighborhood[33] = 255;
        	neighborhood[34] = 255;
        	
        	neighborhood[35] = 255;
        	neighborhood[35] = 255;
        	neighborhood[37] = 255;
        	neighborhood[38] = 255;
        	neighborhood[39] = 255;
        	neighborhood[40] = 255;
        	neighborhood[41] = 255;
        	
        	neighborhood[42] = 255;
        	neighborhood[43] = 255;
        	neighborhood[44] = 255;
        	neighborhood[45] = 255;
        	neighborhood[46] = 255;
        	neighborhood[47] = 255;
        	neighborhood[48] = 255;
	}

	//sort neighborhood
	QuickSort(neighborhood, 0, 7 * 7);
	
	// assign pixel to median

	d_output[yOffset + x] = neighborhood[24];

}

__global__ void medianFilter11( uint8_t *d_input, uint8_t *d_output) {
        // map from threadIdx/BlockIdx to pixel position^M
        int x = blockIdx.x;
        int y = blockIdx.y;
        int dim = 11;

	int yOffsets[11];
	const int yOffset = y * gridDim.x;
	
	yOffsets[0] = yOffset - gridDim.x * 5;
	yOffsets[1] = yOffset - gridDim.x * 4;
	yOffsets[2] = yOffset - gridDim.x * 3;
	yOffsets[3] = yOffset - gridDim.x * 2;
	yOffsets[4] = yOffset - gridDim.x * 1;
	yOffsets[5] = yOffset;
	yOffsets[6] = yOffset + gridDim.x * 1;
	yOffsets[7] = yOffset + gridDim.x * 2;
	yOffsets[8] = yOffset + gridDim.x * 3;
	yOffsets[9] = yOffset + gridDim.x * 4;
	yOffsets[10] = yOffset + gridDim.x * 5;
	
	uint8_t neighborhood[11 * 11];
	
	
	if (y > 0 && y < (gridDim.y - 1) && x > 0 && x < (gridDim.x - 1))
	{
		//for (int i = 0; i < dim; i++)
		{
			for (int j = 0; j < dim; j++)
			{
				for (int k = 0; k < dim / 2; k++)
				{
        				neighborhood[dim * (dim - j - 1) + k] = d_input[yOffsets[j] + x + k];
        				neighborhood[dim * (dim - j - 1) + k + (dim / 2)] = d_input[yOffsets[j] + x - k];
				}
			}
		}
	}
	else
	{
		for (int i = 0; i < 11 * 11 / 2; i++)
		{
			neighborhood[i] = 0;
		}
		neighborhood[60] = d_input[yOffset + x];
		for (int i = 61; i < 11*11; i++)
		{
			neighborhood[i] = 255;
		}
	}

	//sort neighborhood
	QuickSort(neighborhood, 0, 11*11);
	
	// assign pixel to median

	d_output[yOffset + x] = neighborhood[60];

}

__global__ void medianFilter15( uint8_t *d_input, uint8_t *d_output) {
        // map from threadIdx/BlockIdx to pixel position^M
        int x = blockIdx.x;
        int y = blockIdx.y;
        int dim = 15;
	
	const int yOffset = y * gridDim.x;
	
	int yOffsets[7];
	
	yOffsets[0] = yOffset - gridDim.x * 7;
	yOffsets[1] = yOffset - gridDim.x * 6;
	yOffsets[2] = yOffset - gridDim.x * 5;
	yOffsets[3] = yOffset - gridDim.x * 4;
	yOffsets[4] = yOffset - gridDim.x * 3;
	yOffsets[5] = yOffset - gridDim.x * 2;
	yOffsets[6] = yOffset - gridDim.x * 1;
	yOffsets[7] = yOffset;
	yOffsets[8] = yOffset + gridDim.x * 1;
	yOffsets[9] = yOffset + gridDim.x * 2;
	yOffsets[10] = yOffset + gridDim.x * 3;
	yOffsets[11] = yOffset + gridDim.x * 4;
	yOffsets[12] = yOffset + gridDim.x * 5;
	yOffsets[13] = yOffset + gridDim.x * 6;
	yOffsets[14] = yOffset + gridDim.x * 7;
	
	uint8_t neighborhood[7 * 7];
	
	
	if (y > 0 && y < (gridDim.y - 1) && x > 0 && x < (gridDim.x - 1))
	{

        	neighborhood[0] = d_input[yOffsets[0] + x - 7];
        	neighborhood[1] = d_input[yOffsets[0] + x - 6];
        	neighborhood[2] = d_input[yOffsets[0] + x - 5];
        	neighborhood[3] = d_input[yOffsets[0] + x - 4];
        	neighborhood[4] = d_input[yOffsets[0] + x - 3];
        	neighborhood[5] = d_input[yOffsets[0] + x - 2];
        	neighborhood[6] = d_input[yOffsets[0] + x - 1];
        	neighborhood[7] = d_input[yOffsets[0] + x + 0];
        	neighborhood[8] = d_input[yOffsets[0] + x + 1];
        	neighborhood[9] = d_input[yOffsets[0] + x + 2];
        	neighborhood[10] = d_input[yOffsets[0] + x - 3];
        	neighborhood[11] = d_input[yOffsets[0] + x - 4];
        	neighborhood[12] = d_input[yOffsets[0] + x - 5];
        	neighborhood[13] = d_input[yOffsets[0] + x - 6];
        	neighborhood[14] = d_input[yOffsets[0] + x - 7];
        	
        	neighborhood[15] = d_input[yOffsets[1] + x - 7];
        	neighborhood[16] = d_input[yOffsets[1] + x - 6];
        	neighborhood[17] = d_input[yOffsets[1] + x - 5];
        	neighborhood[18] = d_input[yOffsets[1] + x - 4];
        	neighborhood[19] = d_input[yOffsets[1] + x - 3];
        	neighborhood[20] = d_input[yOffsets[1] + x - 2];
        	neighborhood[21] = d_input[yOffsets[1] + x - 1];
        	neighborhood[22] = d_input[yOffsets[1] + x + 0];
        	neighborhood[23] = d_input[yOffsets[1] + x + 1];
        	neighborhood[24] = d_input[yOffsets[1] + x + 2];
        	neighborhood[25] = d_input[yOffsets[1] + x - 3];
        	neighborhood[26] = d_input[yOffsets[1] + x - 4];
        	neighborhood[27] = d_input[yOffsets[1] + x - 5];
        	neighborhood[28] = d_input[yOffsets[1] + x - 6];
        	neighborhood[29] = d_input[yOffsets[1] + x - 7];
        	
        	neighborhood[30] = d_input[yOffsets[2] + x - 7];
        	neighborhood[31] = d_input[yOffsets[2] + x - 6];
        	neighborhood[32] = d_input[yOffsets[2] + x - 5];
        	neighborhood[33] = d_input[yOffsets[2] + x - 4];
        	neighborhood[34] = d_input[yOffsets[2] + x - 3];
        	neighborhood[35] = d_input[yOffsets[2] + x - 2];
        	neighborhood[36] = d_input[yOffsets[2] + x - 1];
        	neighborhood[37] = d_input[yOffsets[2] + x + 0];
        	neighborhood[38] = d_input[yOffsets[2] + x + 1];
        	neighborhood[39] = d_input[yOffsets[2] + x + 2];
        	neighborhood[40] = d_input[yOffsets[2] + x - 3];
        	neighborhood[41] = d_input[yOffsets[2] + x - 4];
        	neighborhood[42] = d_input[yOffsets[2] + x - 5];
        	neighborhood[43] = d_input[yOffsets[2] + x - 6];
        	neighborhood[44] = d_input[yOffsets[2] + x - 7];
        	
        	neighborhood[45] = d_input[yOffsets[3] + x - 7];
        	neighborhood[46] = d_input[yOffsets[3] + x - 6];
        	neighborhood[47] = d_input[yOffsets[3] + x - 5];
        	neighborhood[48] = d_input[yOffsets[3] + x - 4];
        	neighborhood[49] = d_input[yOffsets[3] + x - 3];
        	neighborhood[50] = d_input[yOffsets[3] + x - 2];
        	neighborhood[51] = d_input[yOffsets[3] + x - 1];
        	neighborhood[52] = d_input[yOffsets[3] + x + 0];
        	neighborhood[53] = d_input[yOffsets[3] + x + 1];
        	neighborhood[54] = d_input[yOffsets[3] + x + 2];
        	neighborhood[55] = d_input[yOffsets[3] + x - 3];
        	neighborhood[56] = d_input[yOffsets[3] + x - 4];
        	neighborhood[57] = d_input[yOffsets[3] + x - 5];
        	neighborhood[58] = d_input[yOffsets[3] + x - 6];
        	neighborhood[59] = d_input[yOffsets[3] + x - 7];
        
        	neighborhood[60] = d_input[yOffsets[4] + x - 7];
        	neighborhood[61] = d_input[yOffsets[4] + x - 6];
        	neighborhood[62] = d_input[yOffsets[4] + x - 5];
        	neighborhood[63] = d_input[yOffsets[4] + x - 4];
        	neighborhood[64] = d_input[yOffsets[4] + x - 3];
        	neighborhood[65] = d_input[yOffsets[4] + x - 2];
        	neighborhood[66] = d_input[yOffsets[4] + x - 1];
        	neighborhood[67] = d_input[yOffsets[4] + x + 0];
        	neighborhood[68] = d_input[yOffsets[4] + x + 1];
        	neighborhood[69] = d_input[yOffsets[4] + x + 2];
        	neighborhood[70] = d_input[yOffsets[4] + x - 3];
        	neighborhood[71] = d_input[yOffsets[4] + x - 4];
        	neighborhood[72] = d_input[yOffsets[4] + x - 5];
        	neighborhood[73] = d_input[yOffsets[4] + x - 6];
        	neighborhood[74] = d_input[yOffsets[4] + x - 7];
        	
        	neighborhood[75] = d_input[yOffsets[5] + x - 7];
        	neighborhood[76] = d_input[yOffsets[5] + x - 6];
        	neighborhood[77] = d_input[yOffsets[5] + x - 5];
        	neighborhood[78] = d_input[yOffsets[5] + x - 4];
        	neighborhood[79] = d_input[yOffsets[5] + x - 3];
        	neighborhood[80] = d_input[yOffsets[5] + x - 2];
        	neighborhood[81] = d_input[yOffsets[5] + x - 1];
        	neighborhood[82] = d_input[yOffsets[5] + x + 0];
        	neighborhood[83] = d_input[yOffsets[5] + x + 1];
        	neighborhood[84] = d_input[yOffsets[5] + x + 2];
        	neighborhood[85] = d_input[yOffsets[5] + x - 3];
        	neighborhood[86] = d_input[yOffsets[5] + x - 4];
        	neighborhood[87] = d_input[yOffsets[5] + x - 5];
        	neighborhood[88] = d_input[yOffsets[5] + x - 6];
        	neighborhood[89] = d_input[yOffsets[5] + x - 7];
        	
        	neighborhood[90] = d_input[yOffsets[6] + x - 7];
        	neighborhood[91] = d_input[yOffsets[6] + x - 6];
        	neighborhood[92] = d_input[yOffsets[6] + x - 5];
        	neighborhood[93] = d_input[yOffsets[6] + x - 4];
        	neighborhood[94] = d_input[yOffsets[6] + x - 3];
        	neighborhood[95] = d_input[yOffsets[6] + x - 2];
        	neighborhood[96] = d_input[yOffsets[6] + x - 1];
        	neighborhood[97] = d_input[yOffsets[6] + x + 0];
        	neighborhood[98] = d_input[yOffsets[6] + x + 1];
        	neighborhood[99] = d_input[yOffsets[6] + x + 2];
        	neighborhood[100] = d_input[yOffsets[6] + x - 3];
        	neighborhood[101] = d_input[yOffsets[6] + x - 4];
        	neighborhood[102] = d_input[yOffsets[6] + x - 5];
        	neighborhood[103] = d_input[yOffsets[6] + x - 6];
        	neighborhood[104] = d_input[yOffsets[6] + x - 7];
        	
        	neighborhood[105] = d_input[yOffsets[7] + x - 7];
        	neighborhood[106] = d_input[yOffsets[7] + x - 6];
        	neighborhood[107] = d_input[yOffsets[7] + x - 5];
        	neighborhood[108] = d_input[yOffsets[7] + x - 4];
        	neighborhood[109] = d_input[yOffsets[7] + x - 3];
        	neighborhood[110] = d_input[yOffsets[7] + x - 2];
        	neighborhood[111] = d_input[yOffsets[7] + x - 1];
        	neighborhood[112] = d_input[yOffsets[7] + x + 0];
        	neighborhood[113] = d_input[yOffsets[7] + x + 1];
        	neighborhood[114] = d_input[yOffsets[7] + x + 2];
        	neighborhood[115] = d_input[yOffsets[7] + x - 3];
        	neighborhood[116] = d_input[yOffsets[7] + x - 4];
        	neighborhood[117] = d_input[yOffsets[7] + x - 5];
        	neighborhood[118] = d_input[yOffsets[7] + x - 6];
        	neighborhood[119] = d_input[yOffsets[7] + x - 7];
        	
        	neighborhood[120] = d_input[yOffsets[8] + x - 7];
        	neighborhood[121] = d_input[yOffsets[8] + x - 6];
        	neighborhood[122] = d_input[yOffsets[8] + x - 5];
        	neighborhood[123] = d_input[yOffsets[8] + x - 4];
        	neighborhood[124] = d_input[yOffsets[8] + x - 3];
        	neighborhood[125] = d_input[yOffsets[8] + x - 2];
        	neighborhood[126] = d_input[yOffsets[8] + x - 1];
        	neighborhood[127] = d_input[yOffsets[8] + x + 0];
        	neighborhood[128] = d_input[yOffsets[8] + x + 1];
        	neighborhood[129] = d_input[yOffsets[8] + x + 2];
        	neighborhood[130] = d_input[yOffsets[8] + x - 3];
        	neighborhood[131] = d_input[yOffsets[8] + x - 4];
        	neighborhood[132] = d_input[yOffsets[8] + x - 5];
        	neighborhood[133] = d_input[yOffsets[8] + x - 6];
        	neighborhood[134] = d_input[yOffsets[8] + x - 7];
        	
        	neighborhood[135] = d_input[yOffsets[9] + x - 7];
        	neighborhood[136] = d_input[yOffsets[9] + x - 6];
        	neighborhood[137] = d_input[yOffsets[9] + x - 5];
        	neighborhood[138] = d_input[yOffsets[9] + x - 4];
        	neighborhood[139] = d_input[yOffsets[9] + x - 3];
        	neighborhood[140] = d_input[yOffsets[9] + x - 2];
        	neighborhood[141] = d_input[yOffsets[9] + x - 1];
        	neighborhood[142] = d_input[yOffsets[9] + x + 0];
        	neighborhood[143] = d_input[yOffsets[9] + x + 1];
        	neighborhood[144] = d_input[yOffsets[9] + x + 2];
        	neighborhood[145] = d_input[yOffsets[9] + x - 3];
        	neighborhood[146] = d_input[yOffsets[9] + x - 4];
        	neighborhood[147] = d_input[yOffsets[9] + x - 5];
        	neighborhood[148] = d_input[yOffsets[9] + x - 6];
        	neighborhood[149] = d_input[yOffsets[9] + x - 7];
        	
        	neighborhood[150] = d_input[yOffsets[10] + x - 7];
        	neighborhood[151] = d_input[yOffsets[10] + x - 6];
        	neighborhood[152] = d_input[yOffsets[10] + x - 5];
        	neighborhood[153] = d_input[yOffsets[10] + x - 4];
        	neighborhood[154] = d_input[yOffsets[10] + x - 3];
        	neighborhood[155] = d_input[yOffsets[10] + x - 2];
        	neighborhood[156] = d_input[yOffsets[10] + x - 1];
        	neighborhood[157] = d_input[yOffsets[10] + x + 0];
        	neighborhood[158] = d_input[yOffsets[10] + x + 1];
        	neighborhood[159] = d_input[yOffsets[10] + x + 2];
        	neighborhood[160] = d_input[yOffsets[10] + x - 3];
        	neighborhood[161] = d_input[yOffsets[10] + x - 4];
        	neighborhood[162] = d_input[yOffsets[10] + x - 5];
        	neighborhood[163] = d_input[yOffsets[10] + x - 6];
        	neighborhood[164] = d_input[yOffsets[10] + x - 7];
        
        	neighborhood[165] = d_input[yOffsets[11] + x - 7];
        	neighborhood[166] = d_input[yOffsets[11] + x - 6];
        	neighborhood[167] = d_input[yOffsets[11] + x - 5];
        	neighborhood[167] = d_input[yOffsets[11] + x - 4];
        	neighborhood[169] = d_input[yOffsets[11] + x - 3];
        	neighborhood[170] = d_input[yOffsets[11] + x - 2];
        	neighborhood[171] = d_input[yOffsets[11] + x - 1];
        	neighborhood[172] = d_input[yOffsets[11] + x + 0];
        	neighborhood[173] = d_input[yOffsets[11] + x + 1];
        	neighborhood[174] = d_input[yOffsets[11] + x + 2];
        	neighborhood[175] = d_input[yOffsets[11] + x - 3];
        	neighborhood[176] = d_input[yOffsets[11] + x - 4];
        	neighborhood[177] = d_input[yOffsets[11] + x - 5];
        	neighborhood[178] = d_input[yOffsets[11] + x - 6];
        	neighborhood[179] = d_input[yOffsets[11] + x - 7];
        	
        	neighborhood[180] = d_input[yOffsets[12] + x - 7];
        	neighborhood[181] = d_input[yOffsets[12] + x - 6];
        	neighborhood[182] = d_input[yOffsets[12] + x - 5];
        	neighborhood[183] = d_input[yOffsets[12] + x - 4];
        	neighborhood[184] = d_input[yOffsets[12] + x - 3];
        	neighborhood[185] = d_input[yOffsets[12] + x - 2];
        	neighborhood[186] = d_input[yOffsets[12] + x - 1];
        	neighborhood[187] = d_input[yOffsets[12] + x + 0];
        	neighborhood[188] = d_input[yOffsets[12] + x + 1];
        	neighborhood[189] = d_input[yOffsets[12] + x + 2];
        	neighborhood[190] = d_input[yOffsets[12] + x - 3];
        	neighborhood[191] = d_input[yOffsets[12] + x - 4];
        	neighborhood[192] = d_input[yOffsets[12] + x - 5];
        	neighborhood[193] = d_input[yOffsets[12] + x - 6];
        	neighborhood[194] = d_input[yOffsets[12] + x - 7];
        	
		neighborhood[195] = d_input[yOffsets[13] + x - 7];
        	neighborhood[196] = d_input[yOffsets[13] + x - 6];
        	neighborhood[197] = d_input[yOffsets[13] + x - 5];
        	neighborhood[198] = d_input[yOffsets[13] + x - 4];
        	neighborhood[199] = d_input[yOffsets[13] + x - 3];
        	neighborhood[200] = d_input[yOffsets[13] + x - 2];
        	neighborhood[201] = d_input[yOffsets[13] + x - 1];
        	neighborhood[202] = d_input[yOffsets[13] + x + 0];
        	neighborhood[203] = d_input[yOffsets[13] + x + 1];
        	neighborhood[204] = d_input[yOffsets[13] + x + 2];
        	neighborhood[205] = d_input[yOffsets[13] + x - 3];
        	neighborhood[206] = d_input[yOffsets[13] + x - 4];
        	neighborhood[207] = d_input[yOffsets[13] + x - 5];
        	neighborhood[208] = d_input[yOffsets[13] + x - 6];
        	neighborhood[209] = d_input[yOffsets[13] + x - 7];
        
        	neighborhood[210] = d_input[yOffsets[14] + x - 7];
        	neighborhood[211] = d_input[yOffsets[14] + x - 6];
        	neighborhood[212] = d_input[yOffsets[14] + x - 5];
        	neighborhood[213] = d_input[yOffsets[14] + x - 4];
        	neighborhood[214] = d_input[yOffsets[14] + x - 3];
        	neighborhood[215] = d_input[yOffsets[14] + x - 2];
        	neighborhood[216] = d_input[yOffsets[14] + x - 1];
        	neighborhood[217] = d_input[yOffsets[14] + x + 0];
        	neighborhood[218] = d_input[yOffsets[14] + x + 1];
        	neighborhood[219] = d_input[yOffsets[14] + x + 2];
        	neighborhood[220] = d_input[yOffsets[14] + x - 3];
        	neighborhood[221] = d_input[yOffsets[14] + x - 4];
        	neighborhood[222] = d_input[yOffsets[14] + x - 5];
        	neighborhood[223] = d_input[yOffsets[14] + x - 6];
        	neighborhood[224] = d_input[yOffsets[14] + x - 7];
        	
        	
	}
	else
	{
        	neighborhood[0] = 0;
        	neighborhood[1] = 0;
        	neighborhood[2] = 0;
        	neighborhood[3] = 0;
        	neighborhood[4] = 0;
        	neighborhood[5] = 0;
        	neighborhood[6] = 0;
        	
		neighborhood[7] =  0;
        	neighborhood[8] =  0;
        	neighborhood[9] =  0;
        	neighborhood[10] = 0;
        	neighborhood[11] = 0;
        	neighborhood[12] = 0;
        	neighborhood[13] = 0;
        	
        	neighborhood[14] = 0;
        	neighborhood[15] = 0;
        	neighborhood[16] = 0;
        	neighborhood[17] = 0;
        	neighborhood[18] = 0;
        	neighborhood[19] = 0;
        	neighborhood[20] = 0;
        	
        	neighborhood[21] = 0;
        	neighborhood[22] = 0;
        	neighborhood[23] = 0;
        	
        	neighborhood[24] = d_input[yOffset + x - 0];
        	
        	neighborhood[25] = 255;
        	neighborhood[26] = 255;
        	neighborhood[27] = 255;
        	
        	neighborhood[28] = 255;
        	neighborhood[29] = 255;
        	neighborhood[30] = 255;
        	neighborhood[31] = 255;
        	neighborhood[32] = 255;
        	neighborhood[33] = 255;
        	neighborhood[34] = 255;
        	
        	neighborhood[35] = 255;
        	neighborhood[35] = 255;
        	neighborhood[37] = 255;
        	neighborhood[38] = 255;
        	neighborhood[39] = 255;
        	neighborhood[40] = 255;
        	neighborhood[41] = 255;
        	
        	neighborhood[42] = 255;
        	neighborhood[43] = 255;
        	neighborhood[44] = 255;
        	neighborhood[45] = 255;
        	neighborhood[46] = 255;
        	neighborhood[47] = 255;
        	neighborhood[48] = 255;
	}

	//sort neighborhood
	QuickSort(neighborhood, 0, 7 * 7);
	
	// assign pixel to median

	d_output[yOffset + x] = neighborhood[24];

}

int main (int argc, char *argv[]) {

    if (argc != 4) // Change me per specs
        return 1;

    int dim = atoi(argv[1]);
	int height, width;
    char magic_number[4], input[10];
    int gray_scale;

    //Reads from argv[1] the input pgm file
    FILE *fp = fopen(argv[2],"r");
    fgets(magic_number, 4, fp);
    magic_number[2] = '\0';
	//read up to 10 characters or new line
    fgets(input, 10, fp);
    height = atoi(input);
    fgets(input, 10, fp);
    width = atoi(input);
    fgets(input, 10, fp);
    gray_scale = atoi(input);
    std::vector<uint8_t> mat(height * width);
    //Populates the arrays grabing each pixel from the image and storing it into the vector.
    for (int i= 0; i < height * width; i++)
        mat[i] = fgetc(fp);

    fclose(fp);

    std::vector<uint8_t> median(height * width);
    uint8_t *d_input, *d_output;
    hipMalloc((void **) &d_input, height * width * sizeof(uint8_t));
    hipMalloc((void **) &d_output, height * width * sizeof(uint8_t));
	//copy the image that we read, into d_input and send it over to the GPU's memory
    hipMemcpy(d_input, &mat[0], height * width * sizeof(uint8_t), hipMemcpyHostToDevice);

    // TODO - Fill median.
	dim3 grid(height, width);

	if (dim == 3)
	{
		medianFilter3<<<grid,1>>>(d_input, d_output);
	}
	else if (dim == 7)
	{
		medianFilter7<<<grid,1>>>(d_input, d_output);
	}
	else if (dim == 11)
	{
		medianFilter11<<<grid,1>>>(d_input, d_output);
	}
	else if (dim == 15)
	{}
	else
	{
		std::cout << "Unsuported Filter Size" << std::endl;
		return 1;
	}
    	hipMemcpy(&median[0], d_output, height * width * sizeof(uint8_t), hipMemcpyDeviceToHost);
    	hipFree(d_input);
    	hipFree(d_output);

    //Writes the new pgm picture
    fp = fopen(argv[3], "w");
    fprintf(fp, "%s\n%d\n%d\n%d\n", magic_number, height, width, gray_scale);
    for (int i=0;i<median.size();i++)
        fputc(median[i], fp);
    fclose(fp);

    return 0;
}

